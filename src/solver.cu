#include "hip/hip_runtime.h"
#include "mesh.h"
#include "constant.h"

__device__ double
source(int i, int j, double t){
    return (j==200)?8*sin((t*1e10*2*pi)):0;
}

//H
__global__ void
updateH(mesh* m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idy<H-1){
		m[k].Hzx = m[k].DHx1*m[k].Hzx - m[k].DHx2 * (m[k+H].Ey - m[k].Ey);
		m[k].Hzy = m[k].DHy1*m[k].Hzy + m[k].DHy2 * (m[k+1].Ex - m[k].Ex);
	}
	//source
	__syncthreads();
}

//E
__global__ void
updateE(mesh *m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W && idx>0 && idy>0 && idy<H){
		m[k].Ex = m[k].CEx1*m[k].Ex + m[k].CEx2 * (m[k].Hzx + m[k].Hzy - m[k-1].Hzx - m[k-1].Hzy);
		m[k].Ey = m[k].CEy1*m[k].Ey - m[k].CEy2 * (m[k].Hzx + m[k].Hzy - m[k-H].Hzx - m[k-H].Hzy);
	}
	//source
	__syncthreads();
}

__global__ void 
updateSource(mesh *m, int W, int H, double time){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	if(idx<W && idx>0 && idy>0 && idy<H){
		double s = source(idx, idy, time);
		//double Hs = source(idx, idy, time);
		//double Es = source(idx, idy, time);
		m[k].Hzy+= m[k].DHx2*	s;
		m[k+1].Ey	+= m[k+1].CEy2*s/(120*pi);
	}
	__syncthreads();
}

#define gridsize 64

extern "C"
void cudaUpdateKernel(mesh* d_m, int Nx, int Ny, double t){
	dim3 dimBlock(gridsize,gridsize);
	dim3 dimGrid(ceil(Nx/gridsize), ceil(Ny/gridsize));
	updateH<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateE<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateSource<<<dimGrid, dimBlock>>>(d_m, Nx, Ny, t);
}

